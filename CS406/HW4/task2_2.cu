#include <iostream>
#include <hip/hip_runtime.h>
#include <cstring>

#define WORD_SIZE 4 //this code only works for 4 letter words

///////////// THERE IS A PERFORMANCE SECTION AT THE BOTTOM ///////


//TODO 0: Write the kernel
__global__ void wordCount(const char* text, const char* words, int* count_out, int text_len, int size) {
  
  int idx = blockIdx.x * blockDim.x + threadIdx.x;
  if (idx >= size) return;

  char word[WORD_SIZE];
  for (int i = 0; i < WORD_SIZE; i++){
    word[i] = words[WORD_SIZE * idx + i];
  }
  
  int count_local = 0;

  for (int i = 0; i < text_len-WORD_SIZE + 1; i++){
    bool flag = true;
    for (int j = 0; j < WORD_SIZE; j++){
      if (text[i+j] !=  word[j]){
        flag = false;
        break;
      }
    }
    if (flag){
      count_local++;
    }
  }
  atomicAdd(&count_out[idx], count_local);
}

int main(int argc, char** argv) {
    const char base_text[1024] = "CUDA is a parallel computing platform and parallel application programming interface model created by Nvidia. CUDA gives developers access to the virtual instruction set and memory of the parallel computational elements in CUDA GPUs for parallel computing and parallel execution.";
    //const char base_text[4096] = "Lorem ipsum dolor sit amet, consectetur adipiscing elit. Phasellus mollis vehicula malesuada. Maecenas est leo, mollis id nunc nec, pulvinar aliquam eros. Cras hendrerit quam et neque blandit, eu rhoncus ipsum venenatis. Vestibulum pretium ipsum ac quam viverra, in rutrum orci finibus. Mauris ac est eleifend odio condimentum dignissim. In sit amet laoreet est. Ut sed eros vitae ex dignissim imperdiet. Suspendisse eget pretium purus. Ut dignissim leo eu tortor dapibus accumsan. Lorem ipsum dolor sit amet, consectetur adipiscing elit. Integer ultricies mollis nulla, fermentum malesuada urna cursus a. Phasellus interdum, urna et viverra ornare, nibh neque iaculis nisl, auctor tincidunt dolor justo ut nibh. Curabitur eleifend in eros ac accumsan. Nullam consequat pellentesque erat, quis lobortis arcu iaculis vitae. Praesent a mi metus. Duis pellentesque malesuada aliquam. Duis viverra vitae nunc ut pharetra. Sed at lobortis ipsum, malesuada maximus odio. Sed consequat arcu id nisi tincidunt elementum ac eget sem. In consectetur velit nec ultrices tempor. Quisque a cursus erat. Vestibulum pharetra, mi eu sollicitudin varius, ipsum nulla tristique erat, ut ornare purus mauris ac enim. Ut ultrices mattis est, sed maximus libero pretium ac. Maecenas dignissim congue tellus ut ullamcorper. Integer eget purus vitae eros luctus euismod. Vestibulum et neque sed lacus hendrerit aliquam. Integer eget diam a nulla mattis sodales. Vestibulum varius ut arcu sed laoreet. Integer egestas ex ac ex sagittis, quis auctor lorem efficitur. Vivamus condimentum, ante at mollis tempus, dui eros aliquam libero, congue posuere odio mauris et lacus. Nulla congue in diam bibendum laoreet. Lorem ipsum dolor sit amet, consectetur adipiscing elit. Morbi vel neque a turpis scelerisque luctus. Proin et ipsum sed nisi tincidunt dapibus nec quis nunc. Nunc eleifend orci vitae iaculis elementum. Vestibulum nisl diam, tempus in pellentesque ac, feugiat non felis. Proin vel sodales quam, vel feugiat erat. Etiam hendrerit at lacus sagittis cursus. Nunc ac lorem tortor. Aenean mollis volutpat faucibus. Cras rutrum nisi quis libero feugiat, sed laoreet mauris luctus. Proin sed sollicitudin mauris. In sit amet pretium metus. Aenean fermentum tempus est ac pellentesque. Nam posuere, enim ut ullamcorper pellentesque, mauris erat condimentum purus, vel lobortis diam justo eu massa. Vivamus egestas ultrices sapien quis molestie. Quisque id arcu nec tellus bibendum gravida. Donec vel euismod tortor, et mollis lectus. Integer a eleifend ligula, eu iaculis diam. Sed in odio quis turpis cursus lacinia. Etiam sit amet fermentum est, vitae dapibus nisl. Integer condimentum urna massa, vitae porta nulla cursus non. Suspendisse potenti. Mauris cursus, diam sed vulputate gravida, est sem tincidunt lectus, vel consequat nibh ipsum nec sem. Sed vestibulum ornare felis eu eleifend. Ut vel placerat augue. Nullam quis tincidunt massa. Fusce elementum euismod dui, et eleifend massa sodales sit amet. Pellentesque quis auctor mauris. Ut id mauris scelerisque, blandit odio non, condimentum lectus. Vivamus auctor justo eu erat rutrum lacinia. Nulla et enim magna. Morbi malesuada risus non pellentesque pellentesque. Proin posuere tellus sit amet lacus commodo, vel pulvinar nisi lobortis. Vivamus tempus suscipit mi a hendrerit.";

    const unsigned int multiplier = atoi(argv[1]); //the base_text is copied this number of times to increase the input size

    //copies the base text to create larger instances -----
    const int base_len = strlen(base_text);
    const int text_len = base_len * multiplier;
    char* h_text = new char[text_len];
    for(int i = 0; i < multiplier; i++) {
      memcpy(h_text + (i * base_len), base_text, base_len);
    }

    // Find all possible word sequences
    char h_words[base_len * WORD_SIZE];
    for (int i = 0; i < base_len; i++){
        for (int j = 0; j < WORD_SIZE; j++){
             h_words[i* WORD_SIZE + j] = base_text[(i+j) % base_len];
        }
    }
    //data is ready in h_text -----------------------------
    int h_counts[base_len];
    memset(&h_counts, 0, base_len * sizeof(int));

    // Device variables 
    char *d_text, *d_words;
    int *d_counts;

    //TODO 1: Allocate memory on the device
    hipMalloc((void **)&d_text, text_len * sizeof(char));
    hipMalloc((void **)&d_words, base_len * WORD_SIZE * sizeof(char));
    hipMalloc((void **)&d_counts, base_len * sizeof(int));
    
    //TODO 2: Copy data from host to device
    hipMemcpy(d_text, h_text, text_len * sizeof(char), hipMemcpyHostToDevice);
    hipMemcpy(d_words, h_words, base_len * WORD_SIZE * sizeof(char), hipMemcpyHostToDevice);
    hipMemcpy(d_counts, &h_counts, base_len * sizeof(int), hipMemcpyHostToDevice);

    //TODO 3: Launch the kernel
    wordCount<<< (base_len + 63) / 64, 64>>>(d_text, d_words, d_counts, text_len, base_len);
    hipDeviceSynchronize();

    //TODO 4: Copy result back to host
    hipMemcpy(&h_counts, d_counts,  base_len * sizeof(int), hipMemcpyDeviceToHost);

    //Display the result
    char res_word[WORD_SIZE+1];
    res_word[WORD_SIZE] = '\0';
    int res_count = 0;
    for (int i = 0; i < base_len; i++){        
        if (res_count < h_counts[i]){
            res_count = h_counts[i];
            for (int j = 0; j < WORD_SIZE; j++){
                res_word[j] = h_words[i*WORD_SIZE + j];
            }
        }
    }

    std::cout << "The word \"" << res_word << "\" appears " << res_count << " times in the text." << std::endl;

    //TODO 5: Clear the memory on the device
    hipFree(d_text);
    hipFree(d_words);
    hipFree(d_counts);

    return 0;
}


/*

cs406.gorkemyar@nebula:~/HW4$ nvprof ./task2_2 100
==2623291== NVPROF is profiling process 2623291, command: ./task2_2 100
The word " par" appears 500 times in the text.
==2623291== Profiling application: ./task2_2 100
==2623291== Profiling result:
            Type  Time(%)      Time     Calls       Avg       Min       Max  Name
 GPU activities:   99.95%  10.442ms         1  10.442ms  10.442ms  10.442ms  wordCount(char const *, char const *, int*, int, int)
                    0.04%  4.3830us         3  1.4610us     415ns  3.5520us  [CUDA memcpy HtoD]
                    0.01%  1.1520us         1  1.1520us  1.1520us  1.1520us  [CUDA memcpy DtoH]
      API calls:   96.23%  319.27ms         3  106.42ms  6.0700us  319.26ms  cudaMalloc
                    3.15%  10.439ms         1  10.439ms  10.439ms  10.439ms  cudaDeviceSynchronize
                    0.44%  1.4437ms       456  3.1660us     285ns  165.86us  cuDeviceGetAttribute
                    0.08%  279.65us         1  279.65us  279.65us  279.65us  cudaLaunchKernel
                    0.06%  189.02us         3  63.006us  6.5370us  168.69us  cudaFree
                    0.03%  85.176us         4  21.294us  7.2730us  43.025us  cudaMemcpy
                    0.01%  37.806us         4  9.4510us  6.7890us  16.197us  cuDeviceGetName
                    0.01%  23.560us         4  5.8900us  1.7000us  16.130us  cuDeviceGetPCIBusId
                    0.00%  5.0970us         8     637ns     292ns  2.3980us  cuDeviceGet
                    0.00%  2.2210us         4     555ns     480ns     744ns  cuDeviceTotalMem
                    0.00%  1.4980us         3     499ns     328ns     832ns  cuDeviceGetCount
                    0.00%  1.4120us         4     353ns     320ns     405ns  cuDeviceGetUuid
                    0.00%     527ns         1     527ns     527ns     527ns  cuModuleGetLoadingMode

*/