#include <iostream>
#include <hip/hip_runtime.h>
#include <cstring>

#define WORD_SIZE 4 //this code only works for 4 letter words

//TODO 0: Write the kernel
__global__ void wordCount(const char* text, const char* word, int* count, int text_len) {
  int gap = text_len / (gridDim.x * blockDim.x);
  int start = (blockIdx.x * blockDim.x + threadIdx.x) * gap;
  int end = (threadIdx.x == blockDim.x - 1) && (blockIdx.x == gridDim.x -1) ? text_len : start + gap; 
  int count_local = 0;
  for (int i = start; i < end; i++){
    bool flag = true;
    for (int j = 0; j < WORD_SIZE; j++){
      if (text[i+j] !=  word[j]){
        flag = false;
        break;
      }
    }
    if (flag){
      count_local += 1;
    }
  }
  atomicAdd(count, count_local);

}

int main(int argc, char** argv) {
    const char base_text[1024] = "CUDA is a parallel computing platform and application programming interface model created by Nvidia. CUDA gives developers access to the virtual instruction set and memory of the parallel computational elements in CUDA GPUs.";

    const char* h_word = argv[1]; //word to be searched
    const unsigned int multiplier = atoi(argv[2]); //the base_text is copied this number of times to increase the input size

    //copies the base text to create larger instances -----
    const int base_len = strlen(base_text);
    const int text_len = base_len * multiplier;
    char* h_text = new char[text_len];
    for(int i = 0; i < multiplier; i++) {
      memcpy(h_text + (i * base_len), base_text, base_len);
    }
    //data is ready in h_text -----------------------------
    
    int h_count = 0; //result variable on the host
    // Device variables 
    char *d_text, *d_word;
    int *d_count;

    //TODO 1: Allocate memory on the device
    hipMalloc((void **)&d_text, text_len * sizeof(char));
    hipMalloc((void **)&d_word, WORD_SIZE * sizeof(char));
    hipMalloc((void **)&d_count, sizeof(int));
    
    //TODO 2: Copy data from host to device
    hipMemcpy(d_text, h_text, text_len * sizeof(char), hipMemcpyHostToDevice);
    hipMemcpy(d_word, h_word, WORD_SIZE * sizeof(char), hipMemcpyHostToDevice);
    hipMemcpy(d_count, &h_count, sizeof(int), hipMemcpyHostToDevice);

    //TODO 3: Launch the kernel
    wordCount<<< (multiplier+63) / 64, 64 >>>(d_text, d_word, d_count, text_len);
    hipDeviceSynchronize();
    
    //TODO 4: Copy result back to host
    hipMemcpy(&h_count, d_count,  sizeof(int), hipMemcpyDeviceToHost);

    //Display the result
    std::cout << "The word \"" << h_word << "\" appears " << h_count << " times in the text." << std::endl;

    //TODO 5: Clear the memory on the device
    hipFree(d_text);
    hipFree(d_word);
    hipFree(d_count);

    return 0;
}
