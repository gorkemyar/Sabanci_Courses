#include <iostream>
#include <hip/hip_runtime.h>
#include <cstring>

#define WORD_SIZE 4 //this code only works for 4 letter words

///////////// THERE IS A COMMENT SECTION AT THE BOTTOM ///////

//TODO 0: Write the kernel
__global__ void wordCount(const char* text, const char* words, int* count_out, int text_len) {
  __shared__ char word[WORD_SIZE];
  word[threadIdx.x] = words[blockDim.x * blockIdx.x + threadIdx.x];
  __syncthreads();
  
  int gap = text_len / blockDim.x;
  int start = threadIdx.x * gap;
  int count_local = 0;

  for (int i = start; i < start + gap; i++){
    bool flag = true;
    for (int j = 0; j < WORD_SIZE; j++){
      if (text[i+j] !=  word[j]){
        flag = false;
        break;
      }
    }
    if (flag){
      count_local++;
    }
  }
  atomicAdd(&count_out[blockIdx.x], count_local);
}

int main(int argc, char** argv) {
    const char base_text[1024] = "CUDA is a parallel computing platform and parallel application programming interface model created by Nvidia. CUDA gives developers access to the virtual instruction set and memory of the parallel computational elements in CUDA GPUs for parallel computing and parallel execution.";

    const unsigned int multiplier = atoi(argv[1]); //the base_text is copied this number of times to increase the input size

    //copies the base text to create larger instances -----
    const int base_len = strlen(base_text);
    const int text_len = base_len * multiplier;
    char* h_text = new char[text_len];
    for(int i = 0; i < multiplier; i++) {
      memcpy(h_text + (i * base_len), base_text, base_len);
    }

    // Find all possible word sequences
    char h_words[base_len * WORD_SIZE];
    for (int i = 0; i < base_len; i++){
        for (int j = 0; j < WORD_SIZE; j++){
             h_words[i* WORD_SIZE + j] = base_text[(i+j) % base_len];
        }
    }
    //data is ready in h_text -----------------------------
    int h_counts[base_len];
    memset(&h_counts, 0, base_len * sizeof(int));

    // Device variables 
    char *d_text, *d_words;
    int *d_counts;

    //TODO 1: Allocate memory on the device
    hipMalloc((void **)&d_text, text_len * sizeof(char));
    hipMalloc((void **)&d_words, base_len * WORD_SIZE * sizeof(char));
    hipMalloc((void **)&d_counts, base_len * sizeof(int));
    
    //TODO 2: Copy data from host to device
    hipMemcpy(d_text, h_text, text_len * sizeof(char), hipMemcpyHostToDevice);
    hipMemcpy(d_words, h_words, base_len * WORD_SIZE * sizeof(char), hipMemcpyHostToDevice);
    hipMemcpy(d_counts, &h_counts, base_len * sizeof(int), hipMemcpyHostToDevice);

    //TODO 3: Launch the kernel
    wordCount<<< base_len, WORD_SIZE>>>(d_text, d_words, d_counts, text_len);
    hipDeviceSynchronize();

    //TODO 4: Copy result back to host
    hipMemcpy(&h_counts, d_counts,  base_len * sizeof(int), hipMemcpyDeviceToHost);

    //Display the result
    char res_word[WORD_SIZE+1];
    res_word[WORD_SIZE] = '\0';
    int res_count = 0;
    for (int i = 0; i < base_len; i++){        
        if (res_count < h_counts[i]){
            res_count = h_counts[i];
            for (int j = 0; j < WORD_SIZE; j++){
                res_word[j] = h_words[i*WORD_SIZE + j];
            }
        }
    }


    std::cout << "The word \"" << res_word << "\" appears " << res_count << " times in the text." << std::endl;

    //TODO 5: Clear the memory on the device
    hipFree(d_text);
    hipFree(d_words);
    hipFree(d_counts);

    return 0;
}
/*
cs406.gorkemyar@nebula:~/HW4$ nvprof ./task2 100
==2623810== NVPROF is profiling process 2623810, command: ./task2 100
The word " par" appears 500 times in the text.
==2623810== Profiling application: ./task2 100
==2623810== Profiling result:
            Type  Time(%)      Time     Calls       Avg       Min       Max  Name
 GPU activities:   99.51%  1.1350ms         1  1.1350ms  1.1350ms  1.1350ms  wordCount(char const *, char const *, int*, int)
                    0.39%  4.4170us         3  1.4720us     416ns  3.5840us  [CUDA memcpy HtoD]
                    0.10%  1.1520us         1  1.1520us  1.1520us  1.1520us  [CUDA memcpy DtoH]
      API calls:   98.81%  266.50ms         3  88.833ms  6.1120us  266.49ms  cudaMalloc
                    0.54%  1.4461ms       456  3.1710us     279ns  167.29us  cuDeviceGetAttribute
                    0.42%  1.1310ms         1  1.1310ms  1.1310ms  1.1310ms  cudaDeviceSynchronize
                    0.11%  290.38us         1  290.38us  290.38us  290.38us  cudaLaunchKernel
                    0.07%  188.36us         3  62.785us  6.4500us  168.20us  cudaFree
                    0.03%  77.629us         4  19.407us  7.0630us  38.076us  cudaMemcpy
                    0.01%  35.565us         4  8.8910us  6.3740us  15.323us  cuDeviceGetName
                    0.01%  21.121us         4  5.2800us  1.7250us  14.678us  cuDeviceGetPCIBusId
                    0.00%  4.9520us         8     619ns     283ns  2.0950us  cuDeviceGet
                    0.00%  2.1580us         4     539ns     451ns     747ns  cuDeviceTotalMem
                    0.00%  1.6000us         3     533ns     300ns     980ns  cuDeviceGetCount
                    0.00%  1.4480us         4     362ns     300ns     451ns  cuDeviceGetUuid
                    0.00%     692ns         1     692ns     692ns     692ns  cuModuleGetLoadingMode


This version of task 2 is better since multiple threads can search for the same word in the text. Even though the total thread count
for this version 4 times greater than the other one, this version approximately performed 10 times better.
*/