#include <climits>
#include <iostream>
#include <hip/hip_runtime.h>
#include <cstring>
#include "omp.h"
#include <fstream>
#include <chrono>

#include "gpu/helper.cu"
//#include "gpu/skipPerGpu.cu"
#include "cpu/skipPer.cpp"
#include "cpu/skipOrd.cpp"
#include "multigpu/skipPerMultiGpu.cu"
#include "multigpu/parSpaRyserMultiGpu.cu"

int main(int argc, char* argv[]){
    double* matrix;
    int N, nonzeros;
    int t = 16;
    if (argc != 3){
        cout << "There should be exactly 3 parameters." << endl;
        return 0;
    }
    string filename = argv[1];  // Replace with your file name
    ifstream file(filename);

    if (!file.is_open()) {
        std::cerr << "Failed to open the file: " << filename << std::endl;
        return 1;
    }

    file >> N >> nonzeros;
    // cout<<"N is "<<N<<endl;
    matrix = new double[N*N];
    memset(matrix, 0, N*N*sizeof(double));
    
    for (int i = 0; i < nonzeros; i++) {
        int row_id, col_id;
        double nnz_value;
        file >> row_id >> col_id >> nnz_value;
        matrix[row_id* N+ col_id] = nnz_value;
    }
    
    file.close();

    long gpu_count = stol(argv[2]);
    
    //printArray(matrix, N);
    // cout<<"Number of nonzeros: "<<nonzeros<<endl;
    int* rptrs = new int[N+1], *cptrs = new int[N+1], *colids = new int[nonzeros], *rowids = new int[nonzeros];
    double *cvals = new double[nonzeros], *rvals = new double[nonzeros];
    CRS(matrix, rptrs, colids, rvals, N);
    CCS(matrix, cptrs, rowids, cvals, N);

    auto zerochecks = chrono::high_resolution_clock::now();
    bool isZero = rowColCheck(rptrs, colids, N);
    auto zerochecke= chrono::high_resolution_clock::now();
    double dur = (double)chrono::duration_cast<chrono::nanoseconds>(zerochecke - zerochecks).count() / 1000000000.0;
    if (isZero){
        cout<<0<<" "<<dur<<endl;
        return 0;
    }
    
    double *matrix_skipord = skipOrd(matrix, N);
    int* rptrs_skipord = new int[N+1], *cptrs_skipord = new int[N+1], *colids_skipord = new int[nonzeros], *rowids_skipord = new int[nonzeros];
    double *cvals_skipord = new double[nonzeros], *rvals_skipord = new double[nonzeros];
    CRS(matrix_skipord, rptrs_skipord, colids_skipord, rvals_skipord, N);
    CCS(matrix_skipord, cptrs_skipord, rowids_skipord, cvals_skipord, N);


    auto start3 = chrono::high_resolution_clock::now();
    double result3 = SkipPerMultiGpu(matrix_skipord,  rptrs_skipord,  colids_skipord,  rvals_skipord,  cptrs_skipord,  rowids_skipord,  cvals_skipord, nonzeros, N, nonzeros, gpu_count);
    auto end3 = chrono::high_resolution_clock::now();
    // cout << "Result and time of SkipPerMultiGpu with SkipOrd: " << endl;
    cout << result3 << "\t" << (double)chrono::duration_cast<chrono::nanoseconds>(end3 - start3).count() / 1000000000.0 << endl;
}