#ifndef HELPER_GPU
#define HELPER_GPU
#include <string>
#include <iostream>
#include <set>
#include <vector>
#include <algorithm>
#include <queue>
#include <math.h>
#include <stdio.h>
#include <random>
#include <hip/hip_runtime.h>
#include <cstring>

using namespace std;
typedef pair<int, double> num_pair;

__device__ long binaryToGrayGpu(long num) {
    return num ^ (num >> 1l);
}

__device__ long grayToBinaryGpu(long num) {
    long mask;
    for (mask = num >> 1l; mask != 0; mask = mask >> 1){
        num = num ^ mask;
    }
    return num;
}

__device__ long getBitGpu(long num, long position) {
    return (num >> position) & 1l;
}

__device__ long changeBitGpu(long num, long position) {
    return num ^ (1l << position);
}

__device__ long gFunctionGpu(double* mat, long i, long g, int N)
{
    long minVal = LONG_MAX;
    long pow_j = 1;
    long tmp;
    for(int j = 0; j < N; j++){
        if(mat[i*N + j] != 0){   
            tmp = g < pow_j ? pow_j : g + 2*pow_j - ((g-pow_j) % (pow_j*2)); // next(g, j)
            if (tmp < minVal){
                minVal = tmp;
            }
        }
        pow_j *= 2;
    }
    return minVal;
}

__device__ long nextgGpu(long g, double* X, double* mat, int N, long block_dim, int idx)
{
    long maxVal = LONG_MIN;
    long tmp;
    for(int i = 0; i < N; i++){
        if(X[block_dim*i + idx] == 0){
            tmp = gFunctionGpu(mat, i, g, N);
            if (maxVal < tmp){
                maxVal = tmp;
            }
        }

    }
    if(maxVal == LONG_MIN)
        return g+1;
    return maxVal;
}


#endif