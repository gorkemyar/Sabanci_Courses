#include <climits>
#include <iostream>
#include <hip/hip_runtime.h>
#include <cstring>
#include "omp.h"
#include <fstream>
#include <chrono>

#include "gpu/helper.cu"
//#include "gpu/skipPerGpu.cu"
#include "cpu/skipPer.cpp"
#include "cpu/skipOrd.cpp"
#include "multigpu/skipPerMultiGpu.cu"
#include "multigpu/parSpaRyserMultiGpu.cu"

int main(int argc, char* argv[]){
    double* matrix;
    int N, nonzeros;
    int t = 16;
    if (argc != 3){
        cout << "There should be exactly 3 parameters." << endl;
        return 0;
    }
    string filename = argv[1];  // Replace with your file name
    ifstream file(filename);

    if (!file.is_open()) {
        std::cerr << "Failed to open the file: " << filename << std::endl;
        return 1;
    }

    file >> N >> nonzeros;
    // cout<<"N is "<<N<<endl;
    matrix = new double[N*N];
    memset(matrix, 0, N*N*sizeof(double));
    
    for (int i = 0; i < nonzeros; i++) {
        int row_id, col_id;
        double nnz_value;
        file >> row_id >> col_id >> nnz_value;
        matrix[row_id* N+ col_id] = nnz_value;
    }
    
    file.close();

    long gpu_count = stol(argv[2]);
    
    //printArray(matrix, N);
    // cout<<"Number of nonzeros: "<<nonzeros<<endl;
    int* rptrs = new int[N+1], *cptrs = new int[N+1], *colids = new int[nonzeros], *rowids = new int[nonzeros];
    double *cvals = new double[nonzeros], *rvals = new double[nonzeros];
    CRS(matrix, rptrs, colids, rvals, N);
    CCS(matrix, cptrs, rowids, cvals, N);

    auto zerochecks = chrono::high_resolution_clock::now();
    bool isZero = rowColCheck(rptrs, colids, N);
    auto zerochecke= chrono::high_resolution_clock::now();
    double dur = (double)chrono::duration_cast<chrono::nanoseconds>(zerochecke - zerochecks).count() / 1000000000.0;
    if (isZero){
        cout<<0<<" "<<dur<<endl;
        return 0;
    }
    
    double* matrix_sortord = sortOrd(matrix, N, nonzeros);
    int* rptrs_sortord = new int[N+1], *cptrs_sortord = new int[N+1], *colids_sortord = new int[nonzeros], *rowids_sortord = new int[nonzeros];
    double *cvals_sortord = new double[nonzeros], *rvals_sortord = new double[nonzeros];
    CRS(matrix_sortord, rptrs_sortord, colids_sortord, rvals_sortord, N);
    CCS(matrix_sortord, cptrs_sortord, rowids_sortord, cvals_sortord, N);
    

    auto start4 = chrono::high_resolution_clock::now();
    double result4 = ParSpaRyserMultiGpu(matrix_sortord,  rptrs_sortord,  colids_sortord,  rvals_sortord,  cptrs_sortord,  rowids_sortord,  cvals_sortord, nonzeros, N, nonzeros, gpu_count);
    auto end4 = chrono::high_resolution_clock::now();
    // cout << "Result and time of ParSpaRyserMultiGpu with SortOrd: " << endl;
    cout << result4 << "\t" << (double)chrono::duration_cast<chrono::nanoseconds>(end4 - start4).count() / 1000000000.0 << endl;
}