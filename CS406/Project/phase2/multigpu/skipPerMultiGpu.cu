#include "hip/hip_runtime.h"
#include "../gpu/helper.cu"
#include <climits>
#include <iostream>
#include <hip/hip_runtime.h>
#include <cstring>
#include "omp.h"
#include <fstream>

#define SIZE 40
#define NNZ_SIZE 480

using namespace std;

__global__ void calculation_skipper_multi(double* matrix, int* cptrs, int* rowids, double* cvals, double*row_sum, double* d_result, int N, int NNZ, long start, long end, long chunk){
    long id = blockIdx.x * blockDim.x + threadIdx.x;
    long idx = threadIdx.x;
    long block_dim = blockDim.x;

    __shared__ int l_cptrs[SIZE];
    __shared__ int l_rowids[NNZ_SIZE];
    __shared__ double l_cvals[NNZ_SIZE];
    extern __shared__ double shared_x[];
    double* my_x = (double*) shared_x;
    if (threadIdx.x < N){
        l_cptrs[idx] = cptrs[idx];
        for (int i = cptrs[idx]; i < cptrs[idx + 1]; i++){
            l_rowids[i] = rowids[i];
            l_cvals[i] = cvals[i];
        }
    }
    else if (idx == N){
        l_cptrs[N] = cptrs[N];
    }
    __syncthreads();
    //double my_x[SIZE];
    for (int i = 0; i < N; i++){
        my_x[block_dim*i + idx] = row_sum[i];
    }
        

    long b = chunk*id + start;
    if (b > end) return;
    if (id == 0 && start == 0) b+=1;
    long limit = min((long)(id + 1l)*chunk + start, end);

    long gpre = 0 ^ (0 >> 1); // binary to Gray
    long g = b ^ (b >> 1l); // binary to gray
    double my_result = 0.0;
    double s;
    
    for (; b < limit;){
        long grdiff = g ^ gpre;
        int j = 0;
        while (grdiff > 0){
            if (grdiff & 1){
                s = (2 * ((g >> j) & 1l)) - 1;
                for(int ptr = l_cptrs[j]; ptr < l_cptrs[j+1]; ptr++){
                    my_x[block_dim*l_rowids[ptr] + idx] += (s * l_cvals[ptr]);
                }
            }
            grdiff = grdiff >> 1;
            j++;
        }

        double prod = b & 1 ? -1 : 1;
        for(int i = 0; i < N; i++){
            prod *= my_x[block_dim*i + idx];
        }
        
        my_result +=  prod;
        
        gpre = g;
        if (prod == 0){ // if only one of the my_x[i] = 0
            b = nextgGpu(b, my_x, matrix, N, block_dim, idx);
        }else{
            b++;
        }
        g = binaryToGrayGpu(b);
    }
    
    atomicAdd(&d_result[blockIdx.x], my_result);
}

__host__ double SkipPerMultiGpu(double* matrix, int* rptrs, int* colids, double* rvals, int* cptrs, int* rowids, double* cvals, int nonzeros, int N, int NNZ, long gpu_count)
{      
    int block_size = 2048;
    int thread_count = 128;
    double *row_sum;
    double result = 1;
    row_sum = new double[N];
    memset(row_sum, 0, N * sizeof(double));
    for(int i = 0; i < N; i++){
        double sum = 0;
        for(int intp = rptrs[i]; intp < rptrs[i+1]; intp++){
            sum += rvals[intp];
        }
        row_sum[i] = (double)matrix[(i+1)*N - 1] - (((double)sum)/2.0);
        result *= row_sum[i];
    }
     
   
    long loopVariant = 1l << (long)(N-1);
    long sections = (loopVariant + gpu_count - 1) / gpu_count;
    #pragma omp parallel for num_threads(gpu_count)
    for (int i = 0; i < gpu_count; i++){
        
        double pragma_thread_result = 0;
        long start = i * sections, end = min((i+1)*sections, loopVariant);
        hipSetDevice(i);
        
        /// Device Pointers
        int *d_cptrs, *d_rowids;
        double *d_cvals, *d_row_sum, *d_m;
        double *d_result, *h_result = new double[block_size];
        memset(h_result, 0, block_size * sizeof(double));

        // Allocate Memory for Device
        hipMalloc((void **)&d_rowids, nonzeros * sizeof(int));
        hipMalloc((void **)&d_cvals, nonzeros * sizeof(double));
        hipMalloc((void **)&d_cptrs, (N+1) * sizeof(int));
        hipMalloc((void **)&d_m, (N*N) * sizeof(double));
        hipMalloc((void **)&d_result, block_size * sizeof(double));
        hipMalloc(&d_row_sum, N * sizeof(double));
        // Copy Host Memory to Device Memory
        hipMemcpy(d_cptrs, cptrs,  (N+1)* sizeof(int), hipMemcpyHostToDevice);
        hipMemcpy(d_rowids, rowids,  nonzeros * sizeof(int), hipMemcpyHostToDevice);
        hipMemcpy(d_cvals,   cvals,  nonzeros * sizeof(double), hipMemcpyHostToDevice);
        hipMemcpy(d_m, matrix, N*N*sizeof(double), hipMemcpyHostToDevice);
        hipMemcpy(d_result, h_result, block_size * sizeof(double), hipMemcpyHostToDevice);
        hipMemcpy(d_row_sum, row_sum,  N * sizeof(double), hipMemcpyHostToDevice);
        
        long chunk = (sections + (thread_count * block_size) - 1) / (thread_count * block_size);
        long shared_mem_size = thread_count * sizeof(double) * N;
        calculation_skipper_multi<<<block_size, thread_count, shared_mem_size>>>(d_m, d_cptrs,  d_rowids,  d_cvals, d_row_sum, d_result, N, NNZ, start, end, chunk);
        hipDeviceSynchronize();
        
        hipMemcpy(h_result, d_result, block_size*sizeof(double), hipMemcpyDeviceToHost);
        for (int i = 0; i < block_size; i++){
            pragma_thread_result += h_result[i];
        }

        #pragma omp atomic
        result += pragma_thread_result;

        hipFree(d_rowids);
        hipFree(d_cptrs);
        hipFree(d_cvals);
        hipFree(d_result);
        hipFree(d_row_sum);
        hipFree(d_m);
    }

    return result * (4 * (N%2) - 2);;
}
