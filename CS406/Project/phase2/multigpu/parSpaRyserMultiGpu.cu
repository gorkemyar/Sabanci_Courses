#include "hip/hip_runtime.h"
#include "../gpu/helper.cu"
#include <climits>
#include <iostream>
#include <hip/hip_runtime.h>
#include <cstring>
#include "omp.h"
#include <fstream>

#define SIZE_SO 40
#define NNZ_SIZE_SO 480

using namespace std;

__global__ void calculation_parsparyser_multi(double* matrix, int* cptrs, int* rowids, double* cvals, double*row_sum, double* d_result, int N, int NNZ, long start, long end, long chunk){
    long id = blockIdx.x * blockDim.x + threadIdx.x;
    long idx = threadIdx.x;
    long block_dim = blockDim.x;


    __shared__ int l_cptrs[SIZE_SO];
    __shared__ int l_rowids[NNZ_SIZE_SO];
    __shared__ double l_cvals[NNZ_SIZE_SO];
    extern __shared__ double shared_x[];
    double* my_x = (double*) shared_x;
    if (threadIdx.x < N){
        l_cptrs[idx] = cptrs[idx];
        for (int i = cptrs[idx]; i < cptrs[idx + 1]; i++){
            l_rowids[i] = rowids[i];
            l_cvals[i] = cvals[i];
        }
    }
    else if (idx == N){
        l_cptrs[N] = cptrs[N];
    }
    __syncthreads();
    for (int i = 0; i < N; i++){
        my_x[block_dim*i + idx] = row_sum[i];
    }

    long my_start = chunk*id + start;
    if (my_start > end) return;
    if (id == 0 && start == 0) my_start+=1;
    long my_end = min((long)(id + 1l)*chunk + start, end);

    long my_g = binaryToGrayGpu(my_start-1);
    long g_c = my_g;
    long i = 0;
    while (g_c > 0){
        if (g_c & 1l){
            for (int j = l_cptrs[i]; j < l_cptrs[i+1]; j++){
                my_x[block_dim*l_rowids[j] + idx] += l_cvals[j];
            }
        }
        g_c = g_c >> 1;
        i++;
    }

    double my_result = 0;
    double prod = 1;
    int nzeros = 0;
    for (int i = 0; i < N; i++){
        if (my_x[block_dim*i + idx] != 0){
            prod *= my_x[block_dim*i + idx];
        }else{
            nzeros++;
        }
    }

    long b = my_start;
    long my_g_pre = my_g;
    double div;
    while (b < my_end){
        my_g = binaryToGrayGpu(b);
        long xorVAR = my_g ^ my_g_pre;
        long jking = __ffsll(xorVAR) - 1;
        double s = 2 * getBitGpu(my_g, jking) - 1l;
        div = 1;
        for (int k = l_cptrs[jking]; k < l_cptrs[jking+1]; k++){
            if (my_x[block_dim * l_rowids[k] + idx] == 0){
                nzeros--;
                my_x[block_dim * l_rowids[k] + idx] += s * l_cvals[k];
                prod *= my_x[block_dim * l_rowids[k] + idx];
            }else{
                div *= my_x[block_dim * l_rowids[k] + idx];
                my_x[block_dim * l_rowids[k] + idx] += s * l_cvals[k];
                if (my_x[block_dim * l_rowids[k] + idx] == 0){
                    nzeros++;
                }else{
                    prod *= my_x[block_dim * l_rowids[k] + idx];
                }
            }
        }

        prod /= div;

        if (nzeros == 0){
            double sign = b & 1 ? -1 : 1;
            my_result += sign * prod;
        }
        my_g_pre = my_g;
        b++;
    }

    atomicAdd(&d_result[blockIdx.x], my_result);

}

__host__ double ParSpaRyserMultiGpu(double* matrix, int* rptrs, int* colids, double* rvals, int* cptrs, int* rowids, double* cvals, int nonzeros, int N, int NNZ, long gpu_count)
{      
    int block_size = 2048;
    int thread_count = 128;
    double *row_sum;
    double result = 1;
    row_sum = new double[N];
    memset(row_sum, 0, N * sizeof(double));
    for(int i = 0; i < N; i++){
        double sum = 0;
        for(int intp = rptrs[i]; intp < rptrs[i+1]; intp++){
            sum += rvals[intp];
        }
        row_sum[i] = (double)matrix[(i+1)*N - 1] - (((double)sum)/2.0);
        result *= row_sum[i];
    }
     
   
    long loopVariant = 1l << (long)(N-1);
    long sections = (loopVariant + gpu_count - 1) / gpu_count;
    #pragma omp parallel for num_threads(gpu_count)
    for (int i = 0; i < gpu_count; i++){
        
        double pragma_thread_result = 0;
        long start = i * sections, end = min((i+1)*sections, loopVariant);
        hipSetDevice(i);
        
        /// Device Pointers
        int *d_cptrs, *d_rowids;
        double *d_cvals, *d_row_sum, *d_m;
        double *d_result, *h_result = new double[block_size];
        memset(h_result, 0, block_size * sizeof(double));

        // Allocate Memory for Device
        hipMalloc((void **)&d_rowids, nonzeros * sizeof(int));
        hipMalloc((void **)&d_cvals, nonzeros * sizeof(double));
        hipMalloc((void **)&d_cptrs, (N+1) * sizeof(int));
        hipMalloc((void **)&d_m, (N*N) * sizeof(double));
        hipMalloc((void **)&d_result, block_size * sizeof(double));
        hipMalloc(&d_row_sum, N * sizeof(double));
        // Copy Host Memory to Device Memory
        hipMemcpy(d_cptrs, cptrs,  (N+1)* sizeof(int), hipMemcpyHostToDevice);
        hipMemcpy(d_rowids, rowids,  nonzeros * sizeof(int), hipMemcpyHostToDevice);
        hipMemcpy(d_cvals,   cvals,  nonzeros * sizeof(double), hipMemcpyHostToDevice);
        hipMemcpy(d_m, matrix, N*N*sizeof(double), hipMemcpyHostToDevice);
        hipMemcpy(d_result, h_result, block_size * sizeof(double), hipMemcpyHostToDevice);
        hipMemcpy(d_row_sum, row_sum,  N * sizeof(double), hipMemcpyHostToDevice);
        
        long chunk = (sections + (thread_count * block_size) - 1) / (thread_count * block_size);
        long shared_mem_size = thread_count * sizeof(double) * N;
        calculation_parsparyser_multi<<<block_size, thread_count, shared_mem_size>>>(d_m, d_cptrs,  d_rowids,  d_cvals, d_row_sum, d_result, N, NNZ, start, end, chunk);
        hipDeviceSynchronize();
        
        hipMemcpy(h_result, d_result, block_size*sizeof(double), hipMemcpyDeviceToHost);
        for (int i = 0; i < block_size; i++){
            pragma_thread_result += h_result[i];
        }

        #pragma omp atomic
        result += pragma_thread_result;

        hipFree(d_rowids);
        hipFree(d_cptrs);
        hipFree(d_cvals);
        hipFree(d_result);
        hipFree(d_row_sum);
        hipFree(d_m);
    }

    return result * (4 * (N%2) - 2);;
}
